/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */

#include "lj_gpu_memory.h"
#define LJ_GPU_MemoryT LJ_GPU_Memory<numtyp, acctyp>

template <class numtyp, class acctyp>
int LJ_GPU_MemoryT::bytes_per_atom(const int max_nbors) const {
  return atom.bytes_per_atom()+nbor.bytes_per_atom(max_nbors); 
}

template <class numtyp, class acctyp>
bool LJ_GPU_MemoryT::init(const int ij_size, const int ntypes, 
                          double **host_cutsq, double **host_sigma, 
                          double **host_epsilon, double **host_lj1, 
                          double **host_lj2, double **host_lj3, 
                          double **host_lj4, double **host_offset, 
                          double *host_special_lj, const int max_nbors, 
                          const int me, const int nlocal, const int nall) {
  if (allocated)
    clear();
    
  if (me>=gpu.num_devices())
    return false;
  gpu.set(me);
  if (gpu.revision()<1.0)
    return false;  
    
  // Initialize timers for the selected GPU
  time_pair.init();

  // Initialize atom and nbor data
  max_local=static_cast<int>(static_cast<double>(nlocal)*1.10);
  if (max_local==0)
    max_local=1000;
  if (nall<=nlocal)
    max_atoms=max_local*2;
  else
    max_atoms=static_cast<int>(static_cast<double>(nall)*1.10);
  
  if (!atom.init(max_atoms))
    return false;
  if (!nbor.init(ij_size,max_local,max_nbors))
    return false;
  
  // Get a stream for computing pair potentials
  CUDA_SAFE_CALL(hipStreamCreate(&pair_stream));
    
  // Use the write buffer from atom for data initialization
  NVC_HostT &host_write=atom.host_write;
  assert(host_write.numel()>4 && host_write.numel()>ntypes*ntypes*2);

  // Copy data for bonded interactions
  special_lj.safe_alloc(4);
  special_lj.cast_copy(host_special_lj,host_write);

  // Copy sigma, epsilon, and cutsq onto GPU
  sigma.safe_alloc(ntypes,ntypes,sigma_get_texture<numtyp>());
  sigma.cast_copy(host_sigma[0],host_write);
  epsilon.safe_alloc(ntypes,ntypes,epsilon_get_texture<numtyp>());
  epsilon.cast_copy(host_epsilon[0],host_write);
  cutsq.safe_alloc(ntypes,ntypes,cutsq_get_texture<numtyp>());
  cutsq.cast_copy(host_cutsq[0],host_write);

  // If atom type constants fit in shared memory use fast kernel
  int lj_types=ntypes;
  shared_types=false;
  if (lj_types<=MAX_SHARED_TYPES) {
    lj_types=MAX_SHARED_TYPES;
    shared_types=true;
  }
  offset.safe_alloc(lj_types,lj_types,offset_get_texture<numtyp>());
  offset.cast_copy2D(host_offset[0],host_write,ntypes,ntypes);
  double *t1=host_lj1[0];
  double *t2=host_lj2[0];
  for (int i=0; i<ntypes*ntypes; i++) {
    host_write[i*2]=t1[i];
    host_write[i*2+1]=t2[i];
  }
  lj1.safe_alloc(lj_types,lj_types,lj1_get_texture<numtyp>());
  lj1.copy_2Dfrom_host(reinterpret_cast<typename nvc_vec_traits<numtyp>::vec2 *> (host_write.begin()),
                       ntypes,ntypes);
  t1=host_lj3[0];
  t2=host_lj4[0];
  for (int i=0; i<ntypes*ntypes; i++) {
    host_write[i*2]=t1[i];
    host_write[i*2+1]=t2[i];
  }
  lj3.safe_alloc(lj_types,lj_types,lj3_get_texture<numtyp>());
  lj3.copy_2Dfrom_host(reinterpret_cast<typename nvc_vec_traits<numtyp>::vec2 *> (host_write.begin()),
                       ntypes,ntypes);
        
  dev_error.safe_alloc(1);
  dev_error.zero();
    
  allocated=true;
  return true;
}

template <class numtyp, class acctyp>
void LJ_GPU_MemoryT::clear() {
  if (!allocated)
    return;
  allocated=false;
      
  // Check for any pair style specific errors here
  int err_flag;
  dev_error.copy_to_host(&err_flag);
 
  atom.clear();
  nbor.clear();
    
  CUDA_SAFE_CALL(hipStreamDestroy(pair_stream));

  dev_error.clear();
  sigma.clear();
  epsilon.clear();
  special_lj.clear();
  cutsq.clear();
  offset.clear();
  lj1.clear();
  lj3.clear();
}  
 
template <class numtyp, class acctyp>
double LJ_GPU_MemoryT::host_memory_usage() const {
  return atom.host_memory_usage(max_atoms)+nbor.host_memory_usage()+
         sizeof(LJ_GPU_Memory<numtyp,acctyp>);
}

template class LJ_GPU_Memory<PRECISION,ACC_PRECISION>;
