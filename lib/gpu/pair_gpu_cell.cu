#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */

#if defined(__APPLE__)
#if _GLIBCXX_ATOMIC_BUILTINS == 1
#undef _GLIBCXX_ATOMIC_BUILTINS
#endif // _GLIBCXX_ATOMIC_BUILTINS
#endif // __APPLE__

#include <assert.h>
#include "lj_gpu_memory.h"
#include "pair_gpu_cell.h"

static __constant__ float d_boxlo[3];
static __constant__ float d_boxhi[3];
static __constant__ float d_cell_size[1];
static __constant__ float d_skin[1];

void init_cell_list_const(double cell_size, double skin,
			  double *boxlo, double *boxhi)
{
  float cell_size1 = cell_size;
  float skin1 = skin;
  float boxlo1[3], boxhi1[3];
  for (int i = 0; i < 3; i++) {
    boxlo1[i] = boxlo[i];
    boxhi1[i] = boxhi[i];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(d_cell_size), &cell_size1, sizeof(float),   
		     0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_boxlo),     boxlo1,      3*sizeof(float), 
		     0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_boxhi),     boxhi1,      3*sizeof(float), 
		     0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_skin),      &skin1,       sizeof(float),   
		     0, hipMemcpyHostToDevice); 
}

__global__ void kernel_set_cell_list(unsigned int *cell_idx)
{
  unsigned int gid = threadIdx.x + blockIdx.x*blockDim.x;
  cell_idx[gid] = BIG_NUMBER;
}

// build the cell list
__global__ void kernel_build_cell_list(float3 *cell_list, 
				       unsigned int *cell_idx, 
				       int *cell_type, 
				       int *cell_atom,
				       float3 *pos, 
				       int *type, 
				       const int inum, 
				       const int nall,
				       const int cell_size)
{
  unsigned int gid = threadIdx.x + blockIdx.x*blockDim.x;
  float cSize = d_cell_size[0];
  int ncellx = ceil(((d_boxhi[0] - d_boxlo[0]) + 2.0f*cSize) / cSize);
  int ncelly = ceil(((d_boxhi[1] - d_boxlo[1]) + 2.0f*cSize) / cSize);
  int ncellz = ceil(((d_boxhi[2] - d_boxlo[2]) + 2.0f*cSize) / cSize);

  if (gid < nall) {
    float3 p = pos[gid];
    p.x = fmaxf(p.x, d_boxlo[0]-cSize);
    p.x = fminf(p.x, d_boxhi[0]+cSize);
    p.y = fmaxf(p.y, d_boxlo[1]-cSize);
    p.y = fminf(p.y, d_boxhi[1]+cSize);
    p.z = fmaxf(p.z, d_boxlo[2]-cSize);
    p.z = fminf(p.z, d_boxhi[2]+cSize);

    int cell_id = (int)(p.x/cSize + 1.0) + (int)(p.y/cSize + 1.0) * ncellx
    		    + (int)(p.z/cSize + 1.0) * ncellx * ncelly;

    int atom_pos = atomicAdd(&cell_atom[cell_id], 1);
    int pid = cell_id*cell_size + atom_pos;

    cell_list[pid] = pos[gid];
    cell_type[pid] = type[gid];
    cell_idx [pid] = gid;
    
  }
}

__global__ void kernel_test_rebuild(float3 *cell_list, int *cell_atom, int *rebuild)
{

  float cSize = d_cell_size[0];
  int ncellx = ceil(((d_boxhi[0] - d_boxlo[0]) + 2.0f*cSize) / cSize);
  int ncelly = ceil(((d_boxhi[1] - d_boxlo[1]) + 2.0f*cSize) / cSize);
  int ncellz = ceil(((d_boxhi[2] - d_boxlo[2]) + 2.0f*cSize) / cSize);

  // calculate 3D block idx from 2d block
  int bx = blockIdx.x;
  int by = blockIdx.y % ncelly;
  int bz = blockIdx.y / ncelly;

  int tid = threadIdx.x;

  // compute cell idx from 3D block idx
  int cid = bx + INT_MUL(by, ncellx) + INT_MUL(bz, INT_MUL(ncellx,ncelly));
  int pbase = INT_MUL(cid,blockDim.x); // atom position id in cell list

  float skin = d_skin[0];
  float lowx = d_boxlo[0] + (bx-1)*cSize - 0.5*skin;
  float hix  = lowx + cSize + skin;
  float lowy = d_boxlo[1] + (by-1)*cSize - 0.5*skin;
  float hiy  = lowy + cSize + skin;
  float lowz = d_boxlo[2] + (bz-1)*cSize - 0.5*skin;
  float hiz  = lowz + cSize + skin;

  for (int i = tid; i < cell_atom[cid]; i += blockDim.x) {
    int pid = pbase + i;
    float3 p = cell_list[pid];
    p.x = fmaxf(p.x, d_boxlo[0]-cSize);
    p.x = fminf(p.x, d_boxhi[0]+cSize);
    p.y = fmaxf(p.y, d_boxlo[1]-cSize);
    p.y = fminf(p.y, d_boxhi[1]+cSize);
    p.z = fmaxf(p.z, d_boxlo[2]-cSize);
    p.z = fminf(p.z, d_boxhi[2]+cSize);

    if (p.x < lowx || p.x > hix || p.y < lowy || p.y > hiy || p.z < lowz || p.z > hiz) {
      *rebuild = 1;   
    }
  }

}


__global__ void kernel_test_overflow(int *cell_atom, int *overflow, const int ncell)
{
  unsigned int gid = threadIdx.x + blockIdx.x*blockDim.x;

  if (gid < ncell) {
    if (cell_atom[gid] > blockDim.x) 
      *overflow = 1;
  }
}

__global__ void kernel_copy_list(float3 *cell_list, unsigned int *cell_idx, int *cell_atom, float3 *pos)
{
  float cSize = d_cell_size[0];
  int ncellx = ceil(((d_boxhi[0] - d_boxlo[0]) + 2.0f*cSize) / cSize);
  int ncelly = ceil(((d_boxhi[1] - d_boxlo[1]) + 2.0f*cSize) / cSize);
  int ncellz = ceil(((d_boxhi[2] - d_boxlo[2]) + 2.0f*cSize) / cSize);

  // calculate 3D block idx from 2d block
  int bx = blockIdx.x;
  int by = blockIdx.y % ncelly;
  int bz = blockIdx.y / ncelly;

  int tid = threadIdx.x;

  // compute cell idx from 3D block idx
  int cid = bx + INT_MUL(by, ncellx) + INT_MUL(bz, INT_MUL(ncellx,ncelly));
  int pbase = INT_MUL(cid,blockDim.x); // atom position id in cell list

  for (int i = tid; i < cell_atom[cid]; i += blockDim.x) {
    int pid = pbase + i;
    cell_list[pid] = pos[cell_idx[pid]];
  }

}


__global__ void radixSortBlocks(unsigned int *keys, float3 *values1, int *values2, unsigned int nbits, unsigned int startbit); 



#ifdef __DEVICE_EMULATION__
#define __SYNC __syncthreads();
#else
#define __SYNC 
#endif


#define WARP_SIZE 32

template<class T, int maxlevel>
__device__ T scanwarp(T val, T* sData)
{
    // The following is the same as 2 * RadixSort::WARP_SIZE * warpId + threadInWarp = 
    // 64*(threadIdx.x >> 5) + (threadIdx.x & (RadixSort::WARP_SIZE - 1))
    int idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE - 1));
    sData[idx] = 0;
    idx += WARP_SIZE;
    sData[idx] = val;          __SYNC

#ifdef __DEVICE_EMULATION__
	T t = sData[idx -  1]; __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  2];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  4];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  8];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx - 16];   __SYNC 
        sData[idx] += t;       __SYNC
#else
        if (0 <= maxlevel) { sData[idx] += sData[idx - 1]; } __SYNC
        if (1 <= maxlevel) { sData[idx] += sData[idx - 2]; } __SYNC
        if (2 <= maxlevel) { sData[idx] += sData[idx - 4]; } __SYNC
        if (3 <= maxlevel) { sData[idx] += sData[idx - 8]; } __SYNC
        if (4 <= maxlevel) { sData[idx] += sData[idx -16]; } __SYNC
#endif

        return sData[idx] - val;  // convert inclusive -> exclusive
}

__device__ unsigned int scan(unsigned int idata)
{    
    extern  __shared__  unsigned int ptr[];
    
    unsigned int idx = threadIdx.x;
    
    unsigned int val = idata;
    
    val = scanwarp<unsigned int, 4>(val, ptr);
    __syncthreads();

    if ((idx & (WARP_SIZE - 1)) == WARP_SIZE - 1)
    {
        ptr[idx >> 5] = val + idata;
    }
    __syncthreads();

#ifndef __DEVICE_EMULATION__
    if (idx < WARP_SIZE)
#endif
    {
        ptr[idx] = scanwarp<unsigned int, 2>(ptr[idx], ptr);
    }
    __syncthreads();

    val += ptr[idx >> 5];

    return val;
}


__device__ unsigned int rank(unsigned int preds)
{
    unsigned int address = scan(preds);  

    __shared__ unsigned int numtrue;
    if (threadIdx.x == blockDim.x - 1)
    {
        numtrue = address + preds;
    }
    __syncthreads();

    unsigned int rank;
    unsigned int idx = threadIdx.x;
    rank = (preds) ? address : numtrue + idx - address;

    return rank;
}

template<int blockSize>
__device__ void radixSortBlock(unsigned int *key, float3 *value1, int *value2, unsigned int nbits, unsigned int startbit)
{
  extern __shared__ unsigned int sMem1[];
  __shared__ float sMem2[blockSize];
  __shared__ int sMem3[blockSize];

  int tid = threadIdx.x;

  for(unsigned int shift = startbit; shift < (startbit + nbits); ++shift) {
    unsigned int lsb;
    lsb = !(((*key) >> shift) & 0x1);

    unsigned int r;
		
    r = rank(lsb);

    // This arithmetic strides the ranks across 4 CTA_SIZE regions
    sMem1[r] = *key;
    __syncthreads();

    // The above allows us to read without 4-way bank conflicts:
    *key = sMem1[tid];    
    __syncthreads();

    sMem2[r] = (*value1).x;
    __syncthreads();
    (*value1).x = sMem2[tid];
    __syncthreads();

    sMem2[r] = (*value1).y;
    __syncthreads();
    (*value1).y = sMem2[tid];
    __syncthreads();

    sMem2[r] = (*value1).z;
    __syncthreads();
    (*value1).z = sMem2[tid];
    __syncthreads();

    sMem3[r] = *value2;
    __syncthreads();
    *value2 = sMem3[tid];
    __syncthreads();

  }

}

__global__ void radixSortBlocks(unsigned int *keys, 
				float3 *values1, 
				int *values2, 
				unsigned int nbits, 
				unsigned int startbit)
{

  extern __shared__ unsigned int sMem[];

  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int key;
  float3 value1;
  int value2;
  key = keys[gid];
  value1 = values1[gid];
  value2 = values2[gid];
  __syncthreads();

  if (blockDim.x == 64) 
    radixSortBlock<64>(&key, &value1, &value2, nbits, startbit);
  else if (blockDim.x == 128) 
    radixSortBlock<128>(&key, &value1, &value2, nbits, startbit);
  else if (blockDim.x == 256)
    radixSortBlock<256>(&key, &value1, &value2, nbits, startbit);

  keys[gid] = key;
  values1[gid] = value1;
  values2[gid] = value2;
}

void sortBlocks(unsigned int *keys, float3 *values1, int *values2, const int size, int cell_size)
{
  int i = 0;
  const unsigned int bitSize = sizeof(unsigned int)*8;
  const unsigned int bitStep = 4;
  const int gSize = size/cell_size;
  while (bitSize > i*bitStep) {
    radixSortBlocks<<<gSize, cell_size, 2*cell_size*sizeof(unsigned int)>>>(keys, values1, values2, bitStep, i*bitStep);
    i++;
  }
}

static float3 *d_pos, *pos_temp;
static int *d_type;
static int *d_overflow, *d_rebuild;

void init_cell_list(cell_list &cell_list_gpu, 
		   const int nall,
		   const int ncell, 
		   const int buffer)
{
  hipMalloc((void**)&(cell_list_gpu.pos), ncell*buffer*sizeof(float3));
  hipMalloc((void**)&(cell_list_gpu.idx),  ncell*buffer*sizeof(unsigned int));
  hipMalloc((void**)&(cell_list_gpu.type), ncell*buffer*sizeof(int));
  hipMalloc((void**)&(cell_list_gpu.natom), ncell*sizeof(int));

  hipHostMalloc((void**)&pos_temp, nall*sizeof(float3));
  hipMalloc((void**)&d_pos,       nall*sizeof(float3));
  hipMalloc((void**)&d_type,      nall*sizeof(int));
  hipMalloc((void**)&d_overflow, sizeof(int));
  hipMalloc((void**)&d_rebuild, sizeof(int));

  hipMemset(cell_list_gpu.natom, 0, ncell*sizeof(int));
  hipMemset(cell_list_gpu.pos, 0, ncell*buffer*sizeof(float3));
}

void clear_cell_list(cell_list &cell_list_gpu)
{
  hipFree(cell_list_gpu.pos);
  hipFree(cell_list_gpu.idx);
  hipFree(cell_list_gpu.natom);
  hipFree(cell_list_gpu.type);

  hipHostFree(pos_temp);
  hipFree(d_pos);
  hipFree(d_type);
  hipFree(d_overflow);
  hipFree(d_rebuild);
}


void build_cell_list(double *atom_pos, int *atom_type, 
		     cell_list &cell_list_gpu, 
		     const int ncell, const int ncellx, const int ncelly, const int ncellz, 
		     const int buffer, const int inum, const int nall, const int ago)
{

  hipError_t err;				     

  hipMemset(d_overflow, 0, sizeof(int));
  hipMemset(d_rebuild, 0, sizeof(int));

  // copy position and type to GPU
  for (int i = 0; i < 3*nall; i+=3) { 
    pos_temp[i/3] = make_float3(atom_pos[i], atom_pos[i+1], atom_pos[i+2]);
  }
  hipMemcpy(d_pos, pos_temp, nall*sizeof(float3), hipMemcpyHostToDevice);
  hipMemcpy(d_type, atom_type, nall*sizeof(int),  hipMemcpyHostToDevice);

  static int first_build = 1;
  int rebuild = 0;

  // copy the last built cell-list and test whether it needs to be rebuilt
  if (!first_build) {
    
    dim3 grid(ncellx, ncelly*ncellz);
    kernel_copy_list<<<grid, buffer>>>(cell_list_gpu.pos, 
				 cell_list_gpu.idx, 
				 cell_list_gpu.natom, d_pos);
    hipMemset(d_rebuild, 0, sizeof(int));
    int *temp = (int*)malloc(sizeof(int)*ncell);
    kernel_test_rebuild<<<grid, buffer>>>(cell_list_gpu.pos, 
					 cell_list_gpu.natom,
					 d_rebuild);
    hipMemcpy(&rebuild, d_rebuild, sizeof(int), hipMemcpyDeviceToHost);
    
    err = hipGetLastError();
    assert(err == hipSuccess);
  }

  if (ago == 0) rebuild = 1;
  
  // build cell-list for the first time
  if (first_build || rebuild) {
    first_build = 0;
    // cout << "Building cell list..." << endl;
    hipMemset(cell_list_gpu.natom, 0, ncell*sizeof(int));
    // initialize d_cell_idx for the sorting routine
    kernel_set_cell_list<<<ncell, buffer>>>(cell_list_gpu.idx);
    
    // build cell list
    dim3 blockDim(128);
    dim3 gridDim(static_cast<int>(ceil(static_cast<double>(nall)/blockDim.x)));
    kernel_build_cell_list<<<gridDim, blockDim>>>(cell_list_gpu.pos, 
						  cell_list_gpu.idx, 
						  cell_list_gpu.type, 
						  cell_list_gpu.natom, 
						  d_pos, d_type, inum, nall, buffer);
    err = hipGetLastError();
    assert(err == hipSuccess);
    // check cell list overflow
    int overflow = 0;
    int gDimCell = static_cast<int>(ceil(static_cast<double>(ncell)/buffer));
    kernel_test_overflow<<<gDimCell, buffer>>>(cell_list_gpu.natom, 
					       d_overflow, ncell);
    hipMemcpy(&overflow, d_overflow, sizeof(int), hipMemcpyDeviceToHost);
     
    if (overflow > 0) {
      printf("\n BLOCK_1D too small for cell list, please increase it!");
      printf("\n BLOCK_1D = %d",BLOCK_1D);
      printf("\n ncell = %d",ncell);
      printf("\n gDimCell = %d",gDimCell);
      printf("\n overflow = %d \n",overflow);
      exit(0);
    }
    
    // sort atoms in every cell by atom index to avoid floating point associativity problem.
    sortBlocks(cell_list_gpu.idx, cell_list_gpu.pos, 
	       cell_list_gpu.type, ncell*buffer, buffer);

    hipDeviceSynchronize();
    err = hipGetLastError();
    assert(err == hipSuccess);
  }

}
