/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */

#include "pair_gpu_nbor.h"

int PairGPUNbor::bytes_per_atom(const int max_nbors) const {
  if (_use_packing)
    return (max_nbors*2+4)*sizeof(int);
  else
    return (max_nbors+3)*sizeof(int);
}

bool PairGPUNbor::init(const int ij_size, const int max_atoms, 
                       const int max_nbors) {
  bool success=true;
  if (allocated)
    clear();
    
  // Initialize timers for the selected GPU
  time_nbor.init();

  if (_use_packing)
    success=success && dev_nbor.alloc((max_nbors+4)*max_atoms);
  else  
    success=success && dev_nbor.alloc(3*max_atoms);
  
  success=success && ij.alloc(max_nbors*max_atoms);
  success=success && host_ij.alloc_w(ij_size);
    
  allocated=true;
  
  return success;
}
  
void PairGPUNbor::resize(const int nlocal, const int max_nbor, bool &success) {
  dev_nbor.clear();
  ij.clear();
  if (_use_packing)
    success=success && dev_nbor.alloc((max_nbor+4)*nlocal);
  else  
    success=success && dev_nbor.alloc(3*nlocal);
  success=success && ij.alloc(max_nbor*nlocal);
  allocated=true;
}

void PairGPUNbor::clear() {
  if (!allocated)
    return;
  allocated=false;

  ij.clear();
  host_ij.clear();
  dev_nbor.clear();
}  

double PairGPUNbor::host_memory_usage() const {
  return IJ_SIZE*sizeof(int)+sizeof(PairGPUNbor);
}

void PairGPUNbor::reset(const int inum, int *ilist, const int *numj, 
                        hipStream_t &s) {  
  ij_total=0;

  dev_nbor.copy_from_host(ilist,inum);
  int acc=0;
   
  int ij_size=host_ij.numel();
  if (inum*2<ij_size) {
    for (int i=0; i<inum; i++) {
      host_ij[i]=numj[ilist[i]];
      host_ij[i+inum]=acc;
      acc+=numj[ilist[i]];
    }
    host_ij.copy_to_device(dev_nbor.begin()+inum,2*inum, s);
  } else {
    int offset=0;
    int half=ij_size/2;
    int hi=0;
    for (int i=0; i<inum; i++) {
      host_ij[hi]=numj[ilist[i]];
      host_ij[hi+half]=acc;
      acc+=numj[ilist[i]];
      hi++;
      if (hi==half) {
        host_ij.copy_to_device(dev_nbor.begin()+inum+offset,half,s);
        host_ij.copy_to_device(half,dev_nbor.begin()+2*inum+offset,half,s);
        offset+=half;
        hi=0;
        CUDA_SAFE_CALL(hipStreamSynchronize(s));
      }
   }
   if (hi>0) {
     host_ij.copy_to_device(dev_nbor.begin()+inum+offset,hi,s);
     host_ij.copy_to_device(half,dev_nbor.begin()+2*inum+offset,hi,s);
   }
 }
}
