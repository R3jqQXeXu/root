/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "nvc_macros.h"
#include "nvc_device.h"

// Grabs the properties for all devices
void NVCDevice::init() {
  _properties.clear();
  
  CUDA_SAFE_CALL(hipGetDeviceCount(&_num_devices));
  for (int dev=0; dev<_num_devices; ++dev) {
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
    if (deviceProp.major == 9999 && deviceProp.minor == 9999)
      break;
    _properties.push_back(deviceProp);
  }
  _device=0;
}

// Set the CUDA device to the specified device number
void NVCDevice::set(int num) {
  if (_device==num)
    return;
  hipDeviceReset();
  CUDA_SAFE_CALL(hipSetDevice(num));
  _device=num;
}

// List all devices along with all properties
void NVCDevice::print_all(ostream &out) {
  if (num_devices() == 0)
    printf("There is no device supporting CUDA\n");
  for (int i=0; i<num_devices(); ++i) {
    printf("\nDevice %d: \"%s\"\n", i, name(i).c_str());
    printf("  Revision number:                               %.1f\n", revision(i));
    printf("  Total amount of global memory:                 %.2f GB\n",
           gigabytes(i));
    #if CUDART_VERSION >= 2000
    printf("  Number of multiprocessors:                     %d\n",
           _properties[i].multiProcessorCount);
    printf("  Number of cores:                               %d\n",cores(i));
    #endif
    printf("  Total amount of constant memory:               %u bytes\n",
           _properties[i].totalConstMem); 
    printf("  Total amount of shared memory per block:       %u bytes\n",
           _properties[i].sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n",
           _properties[i].regsPerBlock);
    printf("  Warp size:                                     %d\n",
           _properties[i].warpSize);
    printf("  Maximum number of threads per block:           %d\n",
           _properties[i].maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
           _properties[i].maxThreadsDim[0],
           _properties[i].maxThreadsDim[1],
           _properties[i].maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
           _properties[i].maxGridSize[0],
           _properties[i].maxGridSize[1],
           _properties[i].maxGridSize[2]);
    printf("  Maximum memory pitch:                          %u bytes\n",
           _properties[i].memPitch);
    printf("  Texture alignment:                             %u bytes\n",
           _properties[i].textureAlignment);
    printf("  Clock rate:                                    %.2f GHz\n",
           clock_rate(i));
    #if CUDART_VERSION >= 2000
    printf("  Concurrent copy and execution:                 %s\n",
           _properties[i].deviceOverlap ? "Yes" : "No");
    #endif
  }
}

